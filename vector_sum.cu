/*
  Vector addition
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 128
__global__ void add( int *a, int *b, int *c ) {
  int tid = threadIdx.x;
  if(tid > N-1) return;
  c[tid] = a[tid] + b[tid];
}
int main() {
  int host_a[N], host_b[N], host_c[N];
  int *dev_a, *dev_b, *dev_c;
  for (int i=0; i<N; i++) { host_a[i] = i * i; host_b[i] = - i; }
  hipMalloc( (void**)&dev_a, N * sizeof(int) );
  hipMalloc( (void**)&dev_b, N * sizeof(int) );
  hipMalloc( (void**)&dev_c, N * sizeof(int) );
  hipMemcpy( dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( dev_b, host_b, N * sizeof(int), hipMemcpyHostToDevice );
  add<<<1,N>>>( dev_a, dev_b, dev_c );
  hipMemcpy( host_c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) ;
  for (int i=0; i<N; i++) { printf( "%d + %d = %d\n", host_a[i], host_b[i], host_c[i] ); }
  hipFree( dev_a ); hipFree( dev_b ); hipFree( dev_c );
  return 0;
}