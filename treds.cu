
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
// размер грида
#define DGX 4
#define DGY 8

// размер блока
#define DBX 2
#define DBY 2
#define DBZ 2

// общее количество параллельных процессов: 4*8*2*2*2 = 256
#define N (DBX*DBY*DBZ*DGX*DGY)

__global__ void kern( float *a ) {
  int bs = blockDim.x*blockDim.y*blockDim.z;
  int idx = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*(blockDim.x*blockDim.y) + blockIdx.x*bs + blockIdx.y*bs*gridDim.x ;
  if(idx > N-1) return;
  a[idx] -= 0.5f;
}

int main() {
  float host_a[N], host_c[N];
  float *dev_a;
  srand((unsigned int)time(NULL));
  for(int i=0; i<N; i++) {
    host_a[i] = (float)rand()/(float)RAND_MAX - 0.5f;
  }
  hipMalloc((void**)&dev_a, N * sizeof(float));
  hipMemcpy(dev_a, host_a, N * sizeof(float), hipMemcpyHostToDevice);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  dim3  blocks(DGX,DGY);
  dim3  threads(DBX,DBY,DBZ);
  hipEventRecord(start);
  kern<<<blocks,threads>>>( dev_a );
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipMemcpy(host_c, dev_a, N * sizeof(float), hipMemcpyDeviceToHost);
  for (int i=0; i<N; i++) {
    if(host_a[i]-0.5f != host_c[i]) printf( "[%d]\t %.2f -> %.2f\n",i, host_a[i], host_c[i] );
  }
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "CUDA time simple (ms): " << milliseconds << std::endl;
  hipFree( dev_a ) ; return 0;
}
